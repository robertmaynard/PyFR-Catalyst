#include "PyFRData.h"

#include <algorithm>
#include <cassert>
#include <iostream>
#include <iterator>
#include <limits>
#include <sstream>
#include <stdexcept>

#include <vtkm/CellShape.h>
#include <vtkm/CellTraits.h>
#include <vtkm/TopologyElementTag.h>
#include <vtkm/cont/CellSetSingleType.h>
#include <vtkm/cont/CoordinateSystem.h>
#include <vtkm/cont/DeviceAdapter.h>
#include <vtkm/cont/DataSet.h>
#include <vtkm/cont/Field.h>
#include <vtkm/cont/cuda/DeviceAdapterCuda.h>

#include "ArrayHandleExposed.h"

//------------------------------------------------------------------------------
std::map<int,std::string> PyFRData::fieldName;
std::map<std::string,int> PyFRData::fieldIndex;
bool PyFRData::mapsPopulated = PyFRData::PopulateMaps();

//------------------------------------------------------------------------------
bool PyFRData::PopulateMaps()
{
  fieldName[0] = "density";
  fieldName[1] = "pressure";
  fieldName[2] = "velocity_u";
  fieldName[3] = "velocity_v";
  fieldName[4] = "velocity_w";
  fieldName[5] = "density_gradient_magnitude";
  fieldName[6] = "pressure_gradient_magnitude";
  fieldName[7] = "velocity_gradient_magnitude";
  fieldName[8] = "velocity_qcriterion";

  for (unsigned i=0;i<9;i++)
    fieldIndex[fieldName[i]] = i;

  return true;
}

//------------------------------------------------------------------------------
PyFRData::PyFRData() : catalystData(NULL)
{

}

//------------------------------------------------------------------------------
PyFRData::~PyFRData()
{
}

//------------------------------------------------------------------------------
void PyFRData::Init(void* data)
{
  this->catalystData = static_cast<struct CatalystData*>(data);

  // we only take data from the first stored cell type (i.e. hexahedra)
  MeshDataForCellType* meshData = &(this->catalystData->meshData[0]);
  SolutionDataForCellType* solutionData =
    &(this->catalystData->solutionData[0]);
  this->isovals.resize(this->catalystData->niso);
  std::copy(this->catalystData->isovalues,
            this->catalystData->isovalues+this->catalystData->niso,
            this->isovals.begin());
  std::copy(this->catalystData->eye, this->catalystData->eye+3, this->eye);
  std::copy(this->catalystData->ref, this->catalystData->ref+3, this->ref);
  std::copy(this->catalystData->vup, this->catalystData->vup+3, this->vup);

  typedef ::vtkm::cont::DeviceAdapterTagCuda CudaTag;

  Vec3ArrayHandle vertices;
    {
    const vtkm::Vec<FPType,3> *vecData =
      reinterpret_cast<const vtkm::Vec<FPType,3>*>(meshData->vertices);
    typedef vtkm::cont::internal::Storage<vtkm::Vec<FPType,3>,
                                       vtkm::cont::StorageTagBasic> Vec3Storage;
    Vec3ArrayHandle tmp =
      Vec3ArrayHandle(Vec3Storage(vecData,
                                  meshData->nCells*meshData->nVerticesPerCell));
    vtkm::cont::DeviceAdapterAlgorithm<CudaTag>().
      Copy(tmp, vertices);
    }

  vtkm::cont::ArrayHandle<vtkm::Id> connectivity;
    {
    vtkm::cont::ArrayHandle<int32_t> tmp =
      vtkm::cont::make_ArrayHandle(meshData->con,
                                   (meshData->nSubdividedCells*
                   vtkm::CellTraits<vtkm::CellShapeTagHexahedron>::NUM_POINTS));
    vtkm::cont::ArrayHandleCast<vtkm::Id,
      vtkm::cont::ArrayHandle<int32_t> > cast(tmp);
    vtkm::cont::DeviceAdapterAlgorithm<CudaTag>().
      Copy(cast, connectivity);
    }

  vtkm::cont::CellSetSingleType<> cset(vtkm::CellShapeTagHexahedron(),
                                       meshData->nCells*meshData->nVerticesPerCell,
                                       "cells");
  cset.Fill(connectivity);

  StridedDataFunctor stridedDataFunctor[5];
  for (unsigned i=0;i<5;i++)
    {
    stridedDataFunctor[i].NumberOfCells = meshData->nCells;
    stridedDataFunctor[i].NVerticesPerCell = meshData->nVerticesPerCell;
    stridedDataFunctor[i].NSolutionTypes = 5;
    stridedDataFunctor[i].SolutionType = i;
    stridedDataFunctor[i].CellStride = solutionData->lsdim;
    stridedDataFunctor[i].VertexStride = solutionData->ldim;
    }

  RawDataArrayHandle rawSolutionArray = vtkm::cont::cuda::make_ArrayHandle(
    static_cast<FPType*>(solutionData->solution),
    solutionData->ldim*meshData->nVerticesPerCell);

  DataIndexArrayHandle densityIndexArray(stridedDataFunctor[0],
                                   meshData->nCells*meshData->nVerticesPerCell);
  CatalystMappedDataArrayHandle densityArray(densityIndexArray, rawSolutionArray);

  DataIndexArrayHandle velocity_uIndexArray(stridedDataFunctor[1],
                                   meshData->nCells*meshData->nVerticesPerCell);
  CatalystMappedDataArrayHandle velocity_uArray(velocity_uIndexArray, rawSolutionArray);

  DataIndexArrayHandle velocity_vIndexArray(stridedDataFunctor[2],
                                   meshData->nCells*meshData->nVerticesPerCell);
  CatalystMappedDataArrayHandle velocity_vArray(velocity_vIndexArray, rawSolutionArray);

  DataIndexArrayHandle velocity_wIndexArray(stridedDataFunctor[3],
                                   meshData->nCells*meshData->nVerticesPerCell);
  CatalystMappedDataArrayHandle velocity_wArray(velocity_wIndexArray, rawSolutionArray);

  DataIndexArrayHandle pressureIndexArray(stridedDataFunctor[4],
                                   meshData->nCells*meshData->nVerticesPerCell);
  CatalystMappedDataArrayHandle pressureArray(pressureIndexArray, rawSolutionArray);

  enum ElemType { CONSTANT=0, LINEAR=1, QUADRATIC=2 };
  vtkm::cont::Field density("density",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(densityArray));
  vtkm::cont::Field velocity_u("velocity_u",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(velocity_uArray));
  vtkm::cont::Field velocity_v("velocity_v",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(velocity_vArray));
  vtkm::cont::Field velocity_w("velocity_w",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(velocity_wArray));
  vtkm::cont::Field pressure("pressure",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(pressureArray));

  this->dataSet.AddCoordinateSystem(vtkm::cont::CoordinateSystem("coordinates",
                                                                 1,vertices));
  this->dataSet.AddField(density);
  this->dataSet.AddField(velocity_u);
  this->dataSet.AddField(velocity_v);
  this->dataSet.AddField(velocity_w);
  this->dataSet.AddField(pressure);
  this->dataSet.AddCellSet(cset);
}

//------------------------------------------------------------------------------
void PyFRData::Update()
{
}

bool PyFRData::PrintMetadata() const {
  return this->catalystData->metadata;
}
